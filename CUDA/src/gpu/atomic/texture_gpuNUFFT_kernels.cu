#include "hip/hip_runtime.h"
#ifndef TEXTURE_GPUNUFFT_KERNELS_H
#define TEXTURE_GPUNUFFT_KERNELS_H
#include "gpuNUFFT_kernels.hpp"
#include "../std_gpuNUFFT_kernels.cu"
#include "cuda_utils.cuh"

// ----------------------------------------------------------------------------
// convolutionKernel: NUFFT^H kernel
//
// Performs the gpuNUFFT step by convolution of sample points with
// interpolation function and resampling onto grid. Basic concept based on Zwart
// et al.
//
// parameters:
//  * data           : complex input sample points
//  * crds           : coordinates of data points (x,y,z)
//  * gdata          : output grid data
//  * sectors        : mapping of sample indices according to each sector
//  * sector_centers : coordinates (x,y,z) of sector centers
//  * temp_gdata     : temporary grid data
//  * N              : number of threads
__device__ void textureConvolutionFunction(int *sec, int sec_max,
                                           int sec_offset, DType2 *sdata,
                                           DType2 *data, DType *crds,
                                           CufftType *gdata, IndType *sectors,
                                           IndType *sector_centers)
{
  // start convolution
  int ind, x, y, z;
  int imin, imax, jmin, jmax, kmin, kmax;

  DType dx_sqr, dy_sqr, dz_sqr, val, ix, jy, kz;

  __shared__ IndType3 center;
  center.x = sector_centers[sec[threadIdx.x] * 3];
  center.y = sector_centers[sec[threadIdx.x] * 3 + 1];
  center.z = sector_centers[sec[threadIdx.x] * 3 + 2];

  // Grid Points over Threads
  int data_cnt = sectors[sec[threadIdx.x]] + threadIdx.x + sec_offset;
  // loop over all data points of the current sector, and check if grid position
  // lies inside
  // affected region, if so, add data point weighted to grid position value
  while (data_cnt < sec_max)
  {
    DType3 data_point;  // datapoint per thread
    data_point.x = crds[data_cnt];
    data_point.y = crds[data_cnt + GI.data_count];
    data_point.z = crds[data_cnt + 2 * GI.data_count];

    // set the boundaries of final dataset for gpuNUFFT this point
    ix = mapKSpaceToGrid(data_point.x, GI.gridDims.x, center.x,
                         GI.sector_offset);
    set_minmax(&ix, &imin, &imax, GI.sector_pad_max, GI.kernel_radius);
    jy = mapKSpaceToGrid(data_point.y, GI.gridDims.y, center.y,
                         GI.sector_offset);
    set_minmax(&jy, &jmin, &jmax, GI.sector_pad_max, GI.kernel_radius);
    kz = mapKSpaceToGrid(data_point.z, GI.gridDims.z, center.z,
                         GI.sector_offset);
    set_minmax(&kz, &kmin, &kmax, GI.sector_pad_max, GI.kernel_radius);

    // grid this point onto its cartesian points neighbors
    for (int k = kmin; k <= kmax; k++)
    {
      kz = mapGridToKSpace(k, GI.gridDims.z, center.z, GI.sector_offset);
      dz_sqr = (kz - data_point.z) * GI.aniso_z_scale;
      dz_sqr *= dz_sqr;
      for (int j = jmin; j <= jmax; j++)
      {
        jy = mapGridToKSpace(j, GI.gridDims.y, center.y, GI.sector_offset);
        dy_sqr = (jy - data_point.y) * GI.aniso_y_scale;
        dy_sqr *= dy_sqr;

        for (int i = imin; i <= imax; i++)
        {
          ix = mapGridToKSpace(i, GI.gridDims.x, center.x, GI.sector_offset);
          dx_sqr = (ix - data_point.x) * GI.aniso_x_scale;
          dx_sqr *= dx_sqr;
          // get kernel value
          val = computeTextureLookup(dx_sqr * GI.radiusSquared_inv,
                                     dy_sqr * GI.radiusSquared_inv,
                                     dz_sqr * GI.radiusSquared_inv);

          ind = getIndex(i, j, k, GI.sector_pad_width);

          // multiply data by current kernel val
          // grid complex or scalar
          atomicAdd(&(sdata[ind].x),
              val *
              tex1Dfetch(texDATA, data_cnt).x);

          atomicAdd(&(sdata[ind].y),
              val *
              tex1Dfetch(texDATA, data_cnt).y);
        }  // x
      }  // y
    }  // z
    data_cnt = data_cnt + blockDim.x;
  }  // grid points per sector

  // write shared data to output grid
  __syncthreads();
  // int sector_ind_offset = sec * GI.sector_dim;
  __shared__ int sector_ind_offset;
  sector_ind_offset =
      computeXYZ2Lin(center.x - GI.sector_offset, center.y - GI.sector_offset,
                     center.z - GI.sector_offset, GI.gridDims);

  // each thread writes one position from shared mem to global mem
  for (int s_ind = threadIdx.x; s_ind < GI.sector_dim; s_ind += blockDim.x)
  {
    getCoordsFromIndex(s_ind, &x, &y, &z, GI.sector_pad_width);

    if (isOutlier(x, y, z, center.x, center.y, center.z, GI.gridDims,
                  GI.sector_offset))
      // calculate opposite index
      ind = computeXYZ2Lin(
          calculateOppositeIndex(x, center.x, GI.gridDims.x, GI.sector_offset),
          calculateOppositeIndex(y, center.y, GI.gridDims.y, GI.sector_offset),
          calculateOppositeIndex(z, center.z, GI.gridDims.z, GI.sector_offset),
          GI.gridDims);
    else
      ind = sector_ind_offset +
            computeXYZ2Lin(x, y, z, GI.gridDims);  // index in output grid

    atomicAdd(&(gdata[ind].x), sdata[s_ind].x);  // Re
    atomicAdd(&(gdata[ind].y), sdata[s_ind].y);  // Im
    // reset shared mem
    sdata[s_ind].x = (DType)0.0;
    sdata[s_ind].y = (DType)0.0;
  }
  __syncthreads();
}

__global__ void textureConvolutionKernel(DType2 *data, DType *crds,
                                         CufftType *gdata, IndType *sectors,
                                         IndType *sector_centers, int N)
{
  extern __shared__ DType2 sdata[];  // externally managed shared memory

  // init shared memory
  for (int s_ind = threadIdx.x; s_ind < GI.sector_dim; s_ind += blockDim.x)
  {
    sdata[s_ind].x = (DType)0.0;  // Re
    sdata[s_ind].y = (DType)0.0;  // Im
  }
  __syncthreads();

  __shared__ int sec[THREAD_BLOCK_SIZE];
  sec[threadIdx.x] = blockIdx.x;
  while (sec[threadIdx.x] < N)
  {
    __shared__ int data_max;
    data_max = sectors[sec[threadIdx.x] + 1];
    textureConvolutionFunction(sec, data_max, 0, sdata, data, crds, gdata,
                               sectors, sector_centers);
    __syncthreads();
    sec[threadIdx.x] = sec[threadIdx.x] + gridDim.x;
  }  // sec < sector_count
}

__global__ void balancedTextureConvolutionKernel(
    DType2 *data, DType *crds, CufftType *gdata, IndType *sectors,
    IndType2 *sector_processing_order, IndType *sector_centers, int N)
{
  extern __shared__ DType2 sdata[];  // externally managed shared memory

  // init shared memory
  for (int s_ind = threadIdx.x; s_ind < GI.sector_dim; s_ind += blockDim.x)
  {
    sdata[s_ind].x = (DType)0.0;  // Re
    sdata[s_ind].y = (DType)0.0;  // Im
  }
  __syncthreads();

  int sec_cnt = blockIdx.x;
  __shared__ int sec[THREAD_BLOCK_SIZE];

  while (sec_cnt < N)
  {
    sec[threadIdx.x] = sector_processing_order[sec_cnt].x;
    __shared__ int data_max;
    data_max = min(sectors[sec[threadIdx.x] + 1],
                   sectors[sec[threadIdx.x]] +
                       sector_processing_order[sec_cnt].y + MAXIMUM_PAYLOAD);
    textureConvolutionFunction(sec, data_max,
                               sector_processing_order[sec_cnt].y, sdata, data,
                               crds, gdata, sectors, sector_centers);
    __syncthreads();
    sec_cnt = sec_cnt + gridDim.x;
  }  // sec < sector_count
}

// ----------------------------------------------------------------------------
// convolutionKernel: NUFFT^H kernel
//
// Performs the gpuNUFFT step by convolution of sample points with
// interpolation function and resampling onto grid. Basic concept based on Zwart
// et al.
//
// parameters:
//  * data           : complex input sample points
//  * crds           : coordinates of data points (x,y,z)
//  * gdata          : output grid data
//  * sectors        : mapping of sample indices according to each sector
//  * sector_centers : coordinates (x,y,z) of sector centers
//  * temp_gdata     : temporary grid data
//  * N              : number of threads
__device__ void textureConvolutionFunction2D(DType2 *sdata, int *sec,
                                             int sec_max, int sec_offset,
                                             DType2 *data, DType *crds,
                                             CufftType *gdata, IndType *sectors,
                                             IndType *sector_centers)
{
  // start convolution
  int ind, x, y;
  int imin, imax, jmin, jmax;

  DType dx_sqr, dy_sqr, val, ix, jy;

  __shared__ IndType2 center;
  center.x = sector_centers[sec[threadIdx.x] * 2];
  center.y = sector_centers[sec[threadIdx.x] * 2 + 1];

  // Grid Points over Threads
  int data_cnt = sectors[sec[threadIdx.x]] + threadIdx.x + sec_offset;
  // loop over all data points of the current sector, and check if grid position
  // lies inside
  // affected region, if so, add data point weighted to grid position value
  while (data_cnt < sec_max)
  {
    DType2 data_point;  // datapoint per thread
    data_point.x = crds[data_cnt];
    data_point.y = crds[data_cnt + GI.data_count];

    // set the boundaries of final dataset for gpuNUFFT this point
    ix = mapKSpaceToGrid(data_point.x, GI.gridDims.x, center.x,
                         GI.sector_offset);
    set_minmax(&ix, &imin, &imax, GI.sector_pad_max, GI.kernel_radius);
    jy = mapKSpaceToGrid(data_point.y, GI.gridDims.y, center.y,
                         GI.sector_offset);
    set_minmax(&jy, &jmin, &jmax, GI.sector_pad_max, GI.kernel_radius);

    // grid this point onto its cartesian points neighbors
    for (int j = jmin; j <= jmax; j++)
    {
      jy = mapGridToKSpace(j, GI.gridDims.y, center.y, GI.sector_offset);
      dy_sqr = (jy - data_point.y) * GI.aniso_y_scale;
      dy_sqr *= dy_sqr;

      for (int i = imin; i <= imax; i++)
      {
        ix = mapGridToKSpace(i, GI.gridDims.x, center.x, GI.sector_offset);
        dx_sqr = (ix - data_point.x) * GI.aniso_x_scale;
        dx_sqr *= dx_sqr;
        // get kernel value
        // Calculate Separable Filters
        val = computeTextureLookup(dx_sqr * GI.radiusSquared_inv,
                                   dy_sqr * GI.radiusSquared_inv);

        ind = getIndex2D(i, j, GI.sector_pad_width);

        // multiply data by current kernel val
        // grid complex or scalar
        for (int c = threadIdx.z; c < GI.n_coils_cc; c += blockDim.z)
        {
          atomicAdd(&(sdata[ind + c * GI.sector_dim].x),
                    val * tex1Dfetch(texDATA, data_cnt + c * GI.data_count).x);
          atomicAdd(&(sdata[ind + c * GI.sector_dim].y),
                    val * tex1Dfetch(texDATA, data_cnt + c * GI.data_count).y);
        }
      }  // x
    }  // y
    data_cnt = data_cnt + blockDim.x;
  }  // grid points per sector

  // write shared data to output grid
  __syncthreads();
  // int sector_ind_offset = sec * GI.sector_dim;
  __shared__ int sector_ind_offset;
  sector_ind_offset = computeXY2Lin(center.x - GI.sector_offset,
                                    center.y - GI.sector_offset, GI.gridDims);

  // each thread writes one position from shared mem to global mem
  for (int s_ind = threadIdx.x; s_ind < GI.sector_dim; s_ind += blockDim.x)
  {
    getCoordsFromIndex2D(s_ind, &x, &y, GI.sector_pad_width);

    if (isOutlier2D(x, y, center.x, center.y, GI.gridDims, GI.sector_offset))
      // calculate opposite index
      ind = computeXY2Lin(
          calculateOppositeIndex(x, center.x, GI.gridDims.x, GI.sector_offset),
          calculateOppositeIndex(y, center.y, GI.gridDims.y, GI.sector_offset),
          GI.gridDims);
    else
      ind = sector_ind_offset +
            computeXY2Lin(x, y, GI.gridDims);  // index in output grid

    for (int c = threadIdx.z; c < GI.n_coils_cc; c += blockDim.z)
    {
      atomicAdd(&(gdata[ind + c * GI.gridDims_count].x),
                sdata[s_ind + c * GI.sector_dim].x);  // Re
      atomicAdd(&(gdata[ind + c * GI.gridDims_count].y),
                sdata[s_ind + c * GI.sector_dim].y);  // Im

      // reset shared mem
      sdata[s_ind + c * GI.sector_dim].x = (DType)0.0;
      sdata[s_ind + c * GI.sector_dim].y = (DType)0.0;
    }
  }
}

__global__ void textureConvolutionKernel2D(DType2 *data, DType *crds,
                                           CufftType *gdata, IndType *sectors,
                                           IndType *sector_centers, int N)
{
  extern __shared__ DType2 sdata[];  // externally managed shared memory

  // init shared memory
  for (int s_ind = threadIdx.x; s_ind < GI.sector_dim; s_ind += blockDim.x)
  {
    for (int c = threadIdx.z; c < GI.n_coils_cc; c += blockDim.z)
    {
      sdata[s_ind + c * GI.sector_dim].x = 0.0f;  // Re
      sdata[s_ind + c * GI.sector_dim].y = 0.0f;  // Im
    }
  }
  __syncthreads();

  __shared__ int sec[THREAD_BLOCK_SIZE];
  sec[threadIdx.x] = blockIdx.x;
  while (sec[threadIdx.x] < N)
  {
    __shared__ int data_max;
    data_max = sectors[sec[threadIdx.x] + 1];
    textureConvolutionFunction2D(sdata, sec, data_max, 0, data, crds, gdata,
                                 sectors, sector_centers);
    __syncthreads();
    sec[threadIdx.x] = sec[threadIdx.x] + gridDim.x;
  }  // sec < sector_count
}

__global__ void balancedTextureConvolutionKernel2D(
    DType2 *data, DType *crds, CufftType *gdata, IndType *sectors,
    IndType2 *sector_processing_order, IndType *sector_centers, int N)
{
  extern __shared__ DType2 sdata[];  // externally managed shared memory

  // init shared memory
  for (int s_ind = threadIdx.x; s_ind < GI.sector_dim; s_ind += blockDim.x)
  {
    for (int c = threadIdx.z; c < GI.n_coils_cc; c += blockDim.z)
    {
      sdata[s_ind + c * GI.sector_dim].x = 0.0f;  // Re
      sdata[s_ind + c * GI.sector_dim].y = 0.0f;  // Im
    }
  }
  __syncthreads();

  int sec_cnt = blockIdx.x;
  __shared__ int sec[THREAD_BLOCK_SIZE];

  while (sec_cnt < N)
  {
    sec[threadIdx.x] = sector_processing_order[sec_cnt].x;
    __shared__ int data_max;
    data_max = min(sectors[sec[threadIdx.x] + 1],
                   sectors[sec[threadIdx.x]] 
                      + sector_processing_order[sec_cnt].y + MAXIMUM_PAYLOAD);
    textureConvolutionFunction2D(sdata, sec, data_max,
                                 sector_processing_order[sec_cnt].y, data, crds,
                                 gdata, sectors, sector_centers);
    __syncthreads();
    sec_cnt = sec_cnt + gridDim.x;
  }  // sec < sector_count
}

void performTextureConvolution(DType2 *data_d, DType *crds_d,
                               CufftType *gdata_d, DType *kernel_d,
                               IndType *sectors_d, IndType *sector_centers_d,
                               gpuNUFFT::GpuNUFFTInfo *gi_host)
{
  long shared_mem_size =
      (gi_host->sector_dim) * sizeof(DType2) * gi_host->n_coils_cc;
  int thread_size = THREAD_BLOCK_SIZE;

  dim3 block_dim(thread_size);
  dim3 grid_dim(getOptimalGridDim(gi_host->sector_count, 1));
  if (DEBUG)
  {
    printf("adjoint texture convolution requires %ld bytes of shared memory!\n",
           shared_mem_size);
    printf("grid dim %u, block dim %u \n", grid_dim.x, block_dim.x);
  }
  if (gi_host->is2Dprocessing)
  {
    dim3 block_dim(
        64, 1,
        DEFAULT_VALUE(gi_host->n_coils_cc > 4 ? 4 : gi_host->n_coils_cc));
    textureConvolutionKernel2D <<<grid_dim, block_dim, shared_mem_size>>>
        (data_d, crds_d, gdata_d, sectors_d, sector_centers_d,
         gi_host->sector_count);
  }
  else
    textureConvolutionKernel <<<grid_dim, block_dim, shared_mem_size>>>
        (data_d, crds_d, gdata_d, sectors_d, sector_centers_d,
         gi_host->sector_count);

  if (DEBUG)
    printf("...finished with: %s\n", hipGetErrorString(hipGetLastError()));
}

void performTextureConvolution(DType2 *data_d, DType *crds_d,
                               CufftType *gdata_d, DType *kernel_d,
                               IndType *sectors_d,
                               IndType2 *sector_processing_order_d,
                               IndType *sector_centers_d,
                               gpuNUFFT::GpuNUFFTInfo *gi_host)
{
  long shared_mem_size =
      (gi_host->sector_dim) * sizeof(DType2) * gi_host->n_coils_cc;
  int thread_size = THREAD_BLOCK_SIZE;

  dim3 block_dim(thread_size);
  dim3 grid_dim(getOptimalGridDim(gi_host->sector_count, 1));
  if (DEBUG)
  {
    printf("adjoint balanced texture convolution requires %ld bytes of shared "
           "memory!\n",
           shared_mem_size);
    printf("grid dim %u, block dim %u \n", grid_dim.x, block_dim.x);
  }
  if (gi_host->is2Dprocessing)
  {
    dim3 block_dim(
        64, 1,
        DEFAULT_VALUE(gi_host->n_coils_cc > 4 ? 4 : gi_host->n_coils_cc));
    //printf("block dims: %u %u %u!\n", block_dim.x, block_dim.y, block_dim.z);
    balancedTextureConvolutionKernel2D
            <<<grid_dim, block_dim, shared_mem_size>>>
        (data_d, crds_d, gdata_d, sectors_d, sector_processing_order_d,
         sector_centers_d, gi_host->sectorsToProcess);
  }
  else
    balancedTextureConvolutionKernel <<<grid_dim, block_dim, shared_mem_size>>>
        (data_d, crds_d, gdata_d, sectors_d, sector_processing_order_d,
         sector_centers_d, gi_host->sectorsToProcess);

  if (DEBUG)
    printf("...finished with: %s\n", hipGetErrorString(hipGetLastError()));
}

// ----------------------------------------------------------------------------
// forwardConvolutionKernel: NUFFT kernel
//
// Performs the inverse gpuNUFFT step by convolution of grid points with
// interpolation function and resampling onto trajectory.
//
// parameters:
//  * data           : complex output sample points
//  * crds           : coordinates of data points (x,y,z)
//  * gdata          : input grid data
//  * sectors        : mapping of sample indices according to each sector
//  * sector_centers : coordinates (x,y,z) of sector centers
//  * N              : number of threads

__device__ void
textureForwardConvolutionFunction(long int *sec, long int sec_max, long int sec_offset,
                                  DType2 *sdata, CufftType *gdata_cache,
                                  DType2 *data, DType *crds, CufftType *gdata,
                                  IndType *sectors, IndType *sector_centers)
{
  int ind, imin, imax, jmin, jmax, kmin, kmax, ii, jj, kk;
  DType dx_sqr, dy_sqr, dz_sqr, val, ix, jy, kz;

  __shared__ IndType3 center;
  center.x = sector_centers[sec[threadIdx.x] * 3];
  center.y = sector_centers[sec[threadIdx.x] * 3 + 1];
  center.z = sector_centers[sec[threadIdx.x] * 3 + 2];

  __shared__ long int sector_ind_offset;
  sector_ind_offset =
      computeXYZ2Lin(center.x - GI.sector_offset, center.y - GI.sector_offset,
                     center.z - GI.sector_offset, GI.gridDims);

  // init sector cache
  // preload sector grid data into cache
  for (long int ind = threadIdx.x; ind < GI.sector_dim; ind += blockDim.x)
  {
    long int grid_index;
    getCoordsFromIndex(ind, &ii, &jj, &kk, GI.sector_pad_width);

    if (isOutlier(ii, jj, kk, center.x, center.y, center.z, GI.gridDims,
                  GI.sector_offset))
      // calculate opposite index
      grid_index = computeXYZ2Lin(
          calculateOppositeIndex(ii, center.x, GI.gridDims.x, GI.sector_offset),
          calculateOppositeIndex(jj, center.y, GI.gridDims.y, GI.sector_offset),
          calculateOppositeIndex(kk, center.z, GI.gridDims.z, GI.sector_offset),
          GI.gridDims);
    else
      grid_index = (sector_ind_offset + computeXYZ2Lin(ii, jj, kk, GI.gridDims));

    gdata_cache[ind].x = tex1Dfetch(texGDATA, grid_index).x;
    gdata_cache[ind].y = tex1Dfetch(texGDATA, grid_index).y;
  }

  __syncthreads();

  // Grid Points over Threads
  long int data_cnt = sectors[sec[threadIdx.x]] + threadIdx.x + sec_offset;

  while (data_cnt < sec_max)
  {
    DType3 data_point;  // datapoint per thread
    data_point.x = crds[data_cnt];
    data_point.y = crds[data_cnt + GI.data_count];
    data_point.z = crds[data_cnt + 2 * GI.data_count];

    // set the boundaries of final dataset for gpuNUFFT this point
    ix = mapKSpaceToGrid(data_point.x, GI.gridDims.x, center.x,
                         GI.sector_offset);
    set_minmax(&ix, &imin, &imax, GI.sector_pad_max, GI.kernel_radius);
    jy = mapKSpaceToGrid(data_point.y, GI.gridDims.y, center.y,
                         GI.sector_offset);
    set_minmax(&jy, &jmin, &jmax, GI.sector_pad_max, GI.kernel_radius);
    kz = mapKSpaceToGrid(data_point.z, GI.gridDims.z, center.z,
                         GI.sector_offset);
    set_minmax(&kz, &kmin, &kmax, GI.sector_pad_max, GI.kernel_radius);

    // convolve neighboring cartesian points to this data point
    for (int k = kmin; k <= kmax; k++)
    {
      kz = mapGridToKSpace(k, GI.gridDims.z, center.z, GI.sector_offset);
      dz_sqr = (kz - data_point.z) * GI.aniso_z_scale;
      dz_sqr *= dz_sqr;

      for (int j = jmin; j <= jmax; j++)
      {
        jy = mapGridToKSpace(j, GI.gridDims.y, center.y, GI.sector_offset);
        dy_sqr = (jy - data_point.y) * GI.aniso_y_scale;
        dy_sqr *= dy_sqr;

        for (int i = imin; i <= imax; i++)
        {
          ix = mapGridToKSpace(i, GI.gridDims.x, center.x, GI.sector_offset);
          dx_sqr = (ix - data_point.x) * GI.aniso_x_scale;
          dx_sqr *= dx_sqr;

          // get kernel value
          val = computeTextureLookup(dx_sqr * GI.radiusSquared_inv,
                                     dy_sqr * GI.radiusSquared_inv,
                                     dz_sqr * GI.radiusSquared_inv);

          ind = getIndex(i, j, k, GI.sector_pad_width);

          sdata[threadIdx.x].x += gdata_cache[ind].x * val;
          sdata[threadIdx.x].y += gdata_cache[ind].y * val;
        }  // x loop
      }  // y loop
    }  // z loop
    atomicAdd(&(data[data_cnt].x), sdata[threadIdx.x].x);
    atomicAdd(&(data[data_cnt].y), sdata[threadIdx.x].y);

    data_cnt = data_cnt + blockDim.x;

    sdata[threadIdx.x].x = (DType)0.0;  // Re
    sdata[threadIdx.x].y = (DType)0.0;  // Im
  }  // data points per sector
}

__global__ void textureForwardConvolutionKernel(CufftType *data, DType *crds,
                                                CufftType *gdata,
                                                IndType *sectors,
                                                IndType *sector_centers, int N)
{
  extern __shared__ CufftType shared[];  // externally managed shared memory
  CufftType *shared_out_data = (CufftType *)&shared[0];
  CufftType *gdata_cache = (CufftType *)&shared[blockDim.x];

  __shared__ long int sec[THREAD_BLOCK_SIZE];
  sec[threadIdx.x] = blockIdx.x;

  // init shared memory
  shared_out_data[threadIdx.x].x = (DType)0.0;  // Re
  shared_out_data[threadIdx.x].y = (DType)0.0;  // Im

  __syncthreads();
  // start convolution
  while (sec[threadIdx.x] < N)
  {
    __shared__ long int data_max;
    data_max = sectors[sec[threadIdx.x] + 1];

    textureForwardConvolutionFunction(sec, data_max, 0, shared_out_data,
                                      gdata_cache, data, crds, gdata, sectors,
                                      sector_centers);
    __syncthreads();
    sec[threadIdx.x] = sec[threadIdx.x] + gridDim.x;
  }  // sector check
}

__global__ void balancedTextureForwardConvolutionKernel(
    CufftType *data, DType *crds, CufftType *gdata, IndType *sectors,
    IndType2 *sector_processing_order, IndType *sector_centers, int N)
{
  extern __shared__ CufftType shared[];  // externally managed shared memory
  CufftType *shared_out_data = (CufftType *)&shared[0];
  CufftType *gdata_cache = (CufftType *)&shared[blockDim.x];

  long int sec_cnt = blockIdx.x;
  __shared__ long int sec[THREAD_BLOCK_SIZE];

  // init shared memory
  shared_out_data[threadIdx.x].x = (DType)0.0;  // Re
  shared_out_data[threadIdx.x].y = (DType)0.0;  // Im

  __syncthreads();
  // start convolution
  while (sec_cnt < N)
  {
    sec[threadIdx.x] = sector_processing_order[sec_cnt].x;
    __shared__ long int data_max;
    data_max = min(sectors[sec[threadIdx.x] + 1],
                   sectors[sec[threadIdx.x]] +
                       sector_processing_order[sec_cnt].y + MAXIMUM_PAYLOAD);

    textureForwardConvolutionFunction(
        sec, data_max, sector_processing_order[sec_cnt].y, shared_out_data,
        gdata_cache, data, crds, gdata, sectors, sector_centers);
    __syncthreads();
    sec_cnt = sec_cnt + gridDim.x;
  }  // sector check
}

__device__ void
textureForwardConvolutionFunction2D(int *sec, int sec_max, int sec_offset,
                                    DType2 *sdata, CufftType *gdata_cache,
                                    DType2 *data, DType *crds, CufftType *gdata,
                                    IndType *sectors, IndType *sector_centers)
{
  int ind, imin, imax, jmin, jmax, ii, jj;
  DType val, ix, jy;

  __shared__ IndType2 center;
  center.x = sector_centers[sec[threadIdx.x] * 2];
  center.y = sector_centers[sec[threadIdx.x] * 2 + 1];

  __shared__ int sector_ind_offset;
  sector_ind_offset = computeXY2Lin(center.x - GI.sector_offset,
                                    center.y - GI.sector_offset, GI.gridDims);

  // init sector cache
  // preload sector grid data into cache
  for (int ind = threadIdx.x; ind < GI.sector_dim; ind += blockDim.x)
  {
    int grid_index;
    getCoordsFromIndex2D(ind, &ii, &jj, GI.sector_pad_width);

    // multiply data by current kernel val
    // grid complex or scalar
    if (isOutlier2D(ii, jj, center.x, center.y, GI.gridDims, GI.sector_offset))
      // calculate opposite index
      grid_index = getIndex2D(
          calculateOppositeIndex(ii, center.x, GI.gridDims.x, GI.sector_offset),
          calculateOppositeIndex(jj, center.y, GI.gridDims.y, GI.sector_offset),
          GI.gridDims.x);
    else
      grid_index = (sector_ind_offset + getIndex2D(ii, jj, GI.gridDims.x));

    for (int c = 0; c < GI.n_coils_cc; c++)
    {
      gdata_cache[ind + c * GI.sector_dim].x =
          tex1Dfetch(texGDATA, grid_index + c * GI.gridDims_count).x;
      gdata_cache[ind + c * GI.sector_dim].y =
          tex1Dfetch(texGDATA, grid_index + c * GI.gridDims_count).y;
    }
  }
  __syncthreads();

  // Grid Points over Threads
  int data_cnt = sectors[sec[threadIdx.x]] + threadIdx.x + sec_offset;

  while (data_cnt < sec_max)
  {
    DType2 data_point;  // datapoint per thread
    data_point.x = crds[data_cnt];
    data_point.y = crds[data_cnt + GI.data_count];

    // set the boundaries of final dataset for gpuNUFFT this point
    ix = mapKSpaceToGrid(data_point.x, GI.gridDims.x, center.x,
                         GI.sector_offset);
    set_minmax(&ix, &imin, &imax, GI.sector_pad_max, GI.kernel_radius);
    jy = mapKSpaceToGrid(data_point.y, GI.gridDims.y, center.y,
                         GI.sector_offset);
    set_minmax(&jy, &jmin, &jmax, GI.sector_pad_max, GI.kernel_radius);

    // convolve neighboring cartesian points to this data point
    for (int j = jmin; j <= jmax; j++)
    {
      jy = mapGridToKSpace(j, GI.gridDims.y, center.y, GI.sector_offset);
      DType dy_sqr = (jy - data_point.y) * GI.aniso_y_scale;
      dy_sqr *= dy_sqr;

      for (int i = imin; i <= imax; i++)
      {
        ix = mapGridToKSpace(i, GI.gridDims.x, center.x, GI.sector_offset);
        DType dx_sqr = (ix - data_point.x) * GI.aniso_x_scale;
        dx_sqr *= dx_sqr;
        // get kernel value
        // calc as separable filter
        val = computeTextureLookup(dx_sqr * GI.radiusSquared_inv,
                                   dy_sqr * GI.radiusSquared_inv);

        ind = getIndex2D(i, j, GI.sector_pad_width);

        for (int c = 0; c < GI.n_coils_cc; c++)
        {
          sdata[threadIdx.x + c * blockDim.x].x +=
              gdata_cache[ind + c * GI.sector_dim].x * val;
          sdata[threadIdx.x + c * blockDim.x].y +=
              gdata_cache[ind + c * GI.sector_dim].y * val;
        }
      }  // x loop
    }  // y loop

    for (int c = 0; c < GI.n_coils_cc; c++)
    {
      atomicAdd(&(data[data_cnt + c * GI.data_count].x),
                sdata[threadIdx.x + c * blockDim.x].x);
      atomicAdd(&(data[data_cnt + c * GI.data_count].y),
                sdata[threadIdx.x + c * blockDim.x].y);
      sdata[threadIdx.x + c * blockDim.x].x = (DType)0.0;  // Re
      sdata[threadIdx.x + c * blockDim.x].y = (DType)0.0;  // Im
    }

    data_cnt = data_cnt + blockDim.x;
  }  // data points per sector
}

__device__ void textureForwardConvolutionFunction22D(
    int *sec, int sec_max, int sec_offset, DType2 *data,
    DType *crds, CufftType *gdata, IndType *sectors, IndType *sector_centers)
{
  int imin, imax, jmin, jmax, i, j;
  DType val, ix, jy;

  IndType2 center;
  int sector_ind_offset;
  center.x = sector_centers[sec[threadIdx.x] * 2];
  center.y = sector_centers[sec[threadIdx.x] * 2 + 1];

  sector_ind_offset = computeXY2Lin(center.x - GI.sector_offset,
      center.y - GI.sector_offset, GI.gridDims);

  // Grid Points over Threads
  int data_cnt = sectors[sec[threadIdx.x]] + threadIdx.x + sec_offset;
  __syncthreads();

  while (data_cnt < sec_max)
  {
    DType2 data_point;  // datapoint per thread
    data_point.x = crds[data_cnt];
    data_point.y = crds[data_cnt + GI.data_count];

    // set the boundaries of final dataset for gpuNUFFT this point
    ix = mapKSpaceToGrid(data_point.x, GI.gridDims.x, center.x,
                         GI.sector_offset);
    set_minmax(&ix, &imin, &imax, GI.sector_pad_max, GI.kernel_radius);
    jy = mapKSpaceToGrid(data_point.y, GI.gridDims.y, center.y,
                         GI.sector_offset);
    set_minmax(&jy, &jmin, &jmax, GI.sector_pad_max, GI.kernel_radius);

    // convolve neighboring cartesian points to this data point
    int rangeX = imax - imin + 1;
    int rangeY = jmax - jmin + 1;
    int idx = threadIdx.y;
    int grid_index;

    while (idx < (rangeX * rangeY))
    {
      getCoordsFromIndex2D(idx, &i, &j, rangeX, rangeY);
      i += imin;
      j += jmin;
      if (j <= jmax && j >= jmin)
      {
        jy = mapGridToKSpace(j, GI.gridDims.y, center.y, GI.sector_offset);
        DType dy_sqr = (jy - data_point.y) * GI.aniso_y_scale;
        dy_sqr *= dy_sqr;
        if (i <= imax && i >= imin)
        {
          ix = mapGridToKSpace(i, GI.gridDims.x, center.x, GI.sector_offset);
          DType dx_sqr = (ix - data_point.x) * GI.aniso_x_scale;
          dx_sqr *= dx_sqr;
          // get kernel value
          // calc as separable filter
          val = computeTextureLookup(dx_sqr * GI.radiusSquared_inv,
                                     dy_sqr * GI.radiusSquared_inv);

          if (isOutlier2D(i, j, center.x, center.y, GI.gridDims,
                          GI.sector_offset))
            // calculate opposite index
            grid_index =
                getIndex2D(calculateOppositeIndex(i, center.x, GI.gridDims.x,
                                                  GI.sector_offset),
                           calculateOppositeIndex(j, center.y, GI.gridDims.y,
                                                  GI.sector_offset),
                           GI.gridDims.x);
          else
            grid_index = (sector_ind_offset + getIndex2D(i, j, GI.gridDims.x));

          for (int c = 0; c < GI.n_coils_cc; c++)
          {
            atomicAdd(&(data[data_cnt + c * GI.data_count].x), tex1Dfetch(texGDATA, grid_index + c * GI.gridDims_count).x * val);
            atomicAdd(&(data[data_cnt + c * GI.data_count].y), tex1Dfetch(texGDATA, grid_index + c * GI.gridDims_count).y * val);
          }
        }  // x if
      }    // y if
      idx = idx + blockDim.y;
    }
    data_cnt = data_cnt + blockDim.x;
  }  // data points per sector
}

__device__ void textureForwardConvolutionFunction32D(
    int *sec, int sec_max, int sec_offset, DType *cache, DType2 *data,
    DType *crds, CufftType *gdata, IndType *sectors, IndType *sector_centers)
{
  int imin, imax, jmin, jmax, i, j;
  DType val, ix, jy;

  __shared__ IndType2 center;
  center.x = sector_centers[sec[threadIdx.x] * 2];
  center.y = sector_centers[sec[threadIdx.x] * 2 + 1];

  __shared__ int sector_ind_offset;
  sector_ind_offset = computeXY2Lin(center.x - GI.sector_offset,
      center.y - GI.sector_offset, GI.gridDims);
  int grid_index;

  // Grid Points over Threads
  int data_cnt = sectors[sec[threadIdx.x]] + threadIdx.x + sec_offset;

  while (data_cnt < sec_max)
  {
    DType2 data_point;  // datapoint per thread
    data_point.x = crds[data_cnt];
    data_point.y = crds[data_cnt + GI.data_count];

    // set the boundaries of final dataset for gpuNUFFT this point
    ix = mapKSpaceToGrid(data_point.x, GI.gridDims.x, center.x,
        GI.sector_offset);
    set_minmax(&ix, &imin, &imax, GI.sector_pad_max, GI.kernel_radius);
    jy = mapKSpaceToGrid(data_point.y, GI.gridDims.y, center.y,
        GI.sector_offset);
    set_minmax(&jy, &jmin, &jmax, GI.sector_pad_max, GI.kernel_radius);

    // convolve neighboring cartesian points to this data point
    int idx = threadIdx.y;
    getCoordsFromIndex2D(idx, &i, &j, GI.kernel_width + 1, GI.kernel_width + 1);
    i += imin;
    j += jmin;
    if (j <= jmax && j >= jmin)
    {
      jy = mapGridToKSpace(j, GI.gridDims.y, center.y, GI.sector_offset);
      DType dy_sqr = (jy - data_point.y) * GI.aniso_y_scale;
      dy_sqr *= dy_sqr;
      if (i <= imax && i >= imin)
      {
        ix = mapGridToKSpace(i, GI.gridDims.x, center.x, GI.sector_offset);
        DType dx_sqr = (ix - data_point.x) * GI.aniso_x_scale;
        dx_sqr *= dx_sqr;
        // get kernel value
        // calc as separable filter
        val = computeTextureLookup(dx_sqr * GI.radiusSquared_inv,
            dy_sqr * GI.radiusSquared_inv);
        cache[GI.kernel_widthSquared * threadIdx.x + threadIdx.y] = val;

        if (isOutlier2D(i, j, center.x, center.y, GI.gridDims,
              GI.sector_offset))
          // calculate opposite index
          grid_index =
            getIndex2D(calculateOppositeIndex(i, center.x, GI.gridDims.x,
                  GI.sector_offset),
                calculateOppositeIndex(j, center.y, GI.gridDims.y,
                  GI.sector_offset),
                GI.gridDims.x);
        else
          grid_index = (sector_ind_offset + getIndex2D(i, j, GI.gridDims.x));

        for (int c = 0; c < GI.n_coils_cc; c++)
        {
          atomicAdd(
              &(data[data_cnt + c * GI.data_count].x),
              cache[GI.kernel_widthSquared * threadIdx.x + threadIdx.y] *
              tex1Dfetch(texGDATA, grid_index + c * GI.gridDims_count).x);
          atomicAdd(
              &(data[data_cnt + c * GI.data_count].y),
              cache[GI.kernel_widthSquared * threadIdx.x + threadIdx.y] *
              tex1Dfetch(texGDATA, grid_index + c * GI.gridDims_count).y);
        }
      }  // x if
    }    // y if

    cache[GI.kernel_widthSquared * threadIdx.x + threadIdx.y] = 0;
    data_cnt = data_cnt + blockDim.x;
  }  // data points per sector
}

__global__ void textureForwardConvolutionKernel2D(CufftType *data, DType *crds,
                                                  CufftType *gdata,
                                                  IndType *sectors,
                                                  IndType *sector_centers,
                                                  int N)
{
  extern __shared__ CufftType shared[];  // externally managed shared memory
  CufftType *shared_out_data = (CufftType *)&shared[0];
  CufftType *gdata_cache = (CufftType *)&shared[blockDim.x * GI.n_coils_cc];

  __shared__ int sec[THREAD_BLOCK_SIZE];
  sec[threadIdx.x] = blockIdx.x;

  // init shared memory
  for (int c = 0; c < GI.n_coils_cc; c++)
  {
    shared_out_data[threadIdx.x + c * blockDim.x].x = 0.0f;  // Re
    shared_out_data[threadIdx.x + c * blockDim.x].y = 0.0f;  // Im
  }
  __syncthreads();
  // start convolution
  while (sec[threadIdx.x] < N)
  {
    __shared__ int data_max;
    data_max = sectors[sec[threadIdx.x] + 1];

    textureForwardConvolutionFunction2D(sec, data_max, 0, shared_out_data,
                                        gdata_cache, data, crds, gdata, sectors,
                                        sector_centers);

    __syncthreads();
    sec[threadIdx.x] = sec[threadIdx.x] + gridDim.x;
  }  // sector check
}

__global__ void balancedTextureForwardConvolutionKernel2D(
    CufftType *data, DType *crds, CufftType *gdata, IndType *sectors,
    IndType2 *sector_processing_order, IndType *sector_centers, int N)
{
  extern __shared__ CufftType shared[];  // externally managed shared memory
  CufftType *shared_out_data = (CufftType *)&shared[0];
  CufftType *gdata_cache = (CufftType *)&shared[blockDim.x * GI.n_coils_cc];

  __shared__ int sec[THREAD_BLOCK_SIZE];

  // init shared memory
  for (int c = 0; c < GI.n_coils_cc; c++)
  {
    shared_out_data[threadIdx.x + c * blockDim.x].x = 0.0f;  // Re
    shared_out_data[threadIdx.x + c * blockDim.x].y = 0.0f;  // Im
  }
  __syncthreads();
  // start convolution
  for (int sec_cnt = blockIdx.x; sec_cnt < N; sec_cnt += gridDim.x)
  {
    sec[threadIdx.x] = sector_processing_order[sec_cnt].x;
    __shared__ int data_max;
    data_max = min(sectors[sec[threadIdx.x] + 1],
        sectors[sec[threadIdx.x]] + 
        sector_processing_order[sec_cnt].y + MAXIMUM_PAYLOAD);

    textureForwardConvolutionFunction2D(
        sec, data_max, sector_processing_order[sec_cnt].y, shared_out_data,
        gdata_cache, data, crds, gdata, sectors, sector_centers);

    __syncthreads();
  }  // sector check
}

__global__ void balancedTextureForwardConvolutionKernel22D(
    CufftType *data, DType *crds, CufftType *gdata, IndType *sectors,
    IndType2 *sector_processing_order, IndType *sector_centers, int N)
{
  int sec_cnt = blockIdx.x;
  __shared__ int sec[THREAD_BLOCK_SIZE];

  // init shared memory
  // start convolution
  while (sec_cnt < N)
  {
    int data_max;
    if (threadIdx.y == 0)
    {
      sec[threadIdx.x] = sector_processing_order[sec_cnt].x;
    }
    __syncthreads();

    data_max = min(sectors[sec[threadIdx.x] + 1],
          sectors[sec[threadIdx.x]]
          + sector_processing_order[sec_cnt].y + MAXIMUM_PAYLOAD);

    textureForwardConvolutionFunction22D(
        sec, data_max, sector_processing_order[sec_cnt].y, data, crds,
        gdata, sectors, sector_centers);

    sec_cnt = sec_cnt + gridDim.x;
    __syncthreads();
  }  // sector check
}

__global__ void balancedTextureForwardConvolutionKernel32D(
        CufftType *data, DType *crds, CufftType *gdata, IndType *sectors,
            IndType2 *sector_processing_order, IndType *sector_centers, int N)
{
  extern __shared__ DType shared_cache[];  // externally managed shared memory
  DType *cache = (DType *)&shared_cache[0];

  int sec_cnt = blockIdx.x;
  __shared__ int sec[THREAD_BLOCK_SIZE];

  // init shared memory
  cache[threadIdx.x * blockDim.y + threadIdx.y] = (DType)0.0;
  __syncthreads();
  // start convolution
  while (sec_cnt < N)
  {
      sec[threadIdx.x] = sector_processing_order[sec_cnt].x;
      __shared__ int data_max;
      data_max = min(sectors[sec[threadIdx.x] + 1],
          sectors[sec[threadIdx.x]] + 
          sector_processing_order[sec_cnt].y + MAXIMUM_PAYLOAD);

      textureForwardConvolutionFunction32D(
                  sec, data_max, sector_processing_order[sec_cnt].y, cache, data, crds,
                          gdata, sectors, sector_centers);

      __syncthreads();
      sec_cnt = sec_cnt + gridDim.x;
    }  // sector check
}

void performTextureForwardConvolution(CufftType *data_d, DType *crds_d,
                                      CufftType *gdata_d, DType *kernel_d,
                                      IndType *sectors_d,
                                      IndType *sector_centers_d,
                                      gpuNUFFT::GpuNUFFTInfo *gi_host)
{
  int thread_size = 192;
  long shared_mem_size = (thread_size + gi_host->sector_dim) *
                         gi_host->n_coils_cc * sizeof(CufftType);

  dim3 block_dim(thread_size);
  dim3 grid_dim(getOptimalGridDim(gi_host->sector_count, thread_size));

  if (DEBUG)
    printf("texture forward convolution requires %ld bytes of shared memory!\n",
           shared_mem_size);
  if (gi_host->is2Dprocessing)
  {
    // dim3 block_dim(thread_size, 1, DEFAULT_VALUE(gi_host->n_coils_cc > 4 ? 1
    // : gi_host->n_coils_cc));
    dim3 block_dim(thread_size, 1, 1);  // DEFAULT_VALUE(gi_host->n_coils_cc > 4
                                        // ? 1 : gi_host->n_coils_cc));
    textureForwardConvolutionKernel2D
            <<<grid_dim, block_dim, shared_mem_size>>>
        (data_d, crds_d, gdata_d, sectors_d, sector_centers_d,
         gi_host->sector_count);
  }
  else
    textureForwardConvolutionKernel <<<grid_dim, block_dim, shared_mem_size>>>
        (data_d, crds_d, gdata_d, sectors_d, sector_centers_d,
         gi_host->sector_count);
}

void performTextureForwardConvolution(CufftType *data_d, DType *crds_d,
                                      CufftType *gdata_d, DType *kernel_d,
                                      IndType *sectors_d,
                                      IndType2 *sector_processing_order_d,
                                      IndType *sector_centers_d,
                                      gpuNUFFT::GpuNUFFTInfo *gi_host)
{
  int thread_size = THREAD_BLOCK_SIZE;
  long shared_mem_size = (thread_size + gi_host->sector_dim) *
                         gi_host->n_coils_cc * sizeof(CufftType);

  dim3 block_dim(thread_size);
  dim3 grid_dim(getOptimalGridDim(gi_host->sector_count, thread_size));

  if (DEBUG)
    printf("balanced texture forward convolution requires %ld bytes of shared "
           "memory!\n",
           shared_mem_size);
  if (gi_host->is2Dprocessing)
  {
    bool useV2cached = false;

    if (useV2cached)
    {
      int thread_size = 32;
      int threadY = (gi_host->kernel_width + 1) * (gi_host->kernel_width + 1);

      long shared_mem_size =
        (threadY * thread_size) * sizeof(DType);

      grid_dim = dim3(getOptimalGridDim(gi_host->sector_count, 1));

      block_dim = getOptimal2DBlockDim(thread_size, threadY);

      if (DEBUG)
      {
        printf("balanced texture forward convolution 2 (2d) requires %ld bytes "
            "of shared memory!\n",
            shared_mem_size);
        printf("block dims: %u %u %u!\n", block_dim.x, block_dim.y, block_dim.z);
        printf("grid dims: %u %u %u!\n", grid_dim.x, grid_dim.y, grid_dim.z);
      }

      balancedTextureForwardConvolutionKernel32D<<<grid_dim, block_dim, shared_mem_size>>>
        (data_d, crds_d, gdata_d, sectors_d, sector_processing_order_d, sector_centers_d, gi_host->sectorsToProcess);
    }
    else
    {
      int thread_size = 32;
      long shared_mem_size =
        (gi_host->kernel_widthSquared * thread_size) * sizeof(DType);

      grid_dim = dim3(getOptimalGridDim(gi_host->sector_count, 1));

      //TODO maybe it's better to round kwSqrd to the next multiple of 2
      block_dim = getOptimal2DBlockDim(thread_size, gi_host->kernel_widthSquared);

      if (DEBUG)
      {
        printf("balanced texture forward convolution 2 (2d) requires %ld bytes "
            "of shared memory!\n",
            shared_mem_size);
        printf("grid dims: %u %u %u!\n", grid_dim.x, grid_dim.y, grid_dim.z);
        printf("block dims: %u %u %u!\n", block_dim.x, block_dim.y, block_dim.z);
      }

      balancedTextureForwardConvolutionKernel22D<<<grid_dim, block_dim, shared_mem_size>>>
        (data_d, crds_d, gdata_d, sectors_d, sector_processing_order_d, sector_centers_d, gi_host->sectorsToProcess);
    }
  }
  else
  {
    balancedTextureForwardConvolutionKernel
            <<<grid_dim, block_dim, shared_mem_size>>>
        (data_d, crds_d, gdata_d, sectors_d, sector_processing_order_d,
         sector_centers_d, gi_host->sectorsToProcess);
  }
}

#endif
